#include "hip/hip_runtime.h"
#include "../include/BarnesHut.cuh"

void gpuAssert(hipError_t code, const char *file, int line, bool abort)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) getchar();
    }
}

void CheckCudaCall(hipError_t command, const char * commandName, const char * fileName, int line)
{
    if (command != hipSuccess)
    {
        fprintf(stderr, "Error: CUDA result \"%s\" for call \"%s\" in file \"%s\" at line %d. Terminating...\n",
                hipGetErrorString(command), commandName, fileName, line);
        exit(0);
    }
}

BarnesHut::BarnesHut(const SimulationParameters p) {

    parameters = p;
    KernelHandler = KernelsWrapper(p);
    step = 0;

    h_subDomainHandler = new SubDomainKeyTree();
    /*h_subDomainHandler->rank =*/
    MPI_Comm_rank(MPI_COMM_WORLD, &h_subDomainHandler->rank); //0;
    h_subDomainHandler->range = new unsigned long[3];
    h_subDomainHandler->range[0] = 0;
    h_subDomainHandler->range[1] = 2305843009213693952UL; //4611686018427387904UL;// + 3872UL;
    h_subDomainHandler->range[2] = KEY_MAX;
    /*h_subDomainHandler->numProcesses =*/
    MPI_Comm_size(MPI_COMM_WORLD, &h_subDomainHandler->numProcesses); //2;

    numParticles = p.numberOfParticles; //NUM_BODIES;
    numNodes = 2 * numParticles + 12000; //+ 12000; //2 * numParticles + 12000;
    numParticlesLocal = numParticles/h_subDomainHandler->numProcesses;

    Logger(DEBUG) << "numParticles: " << numParticles << "  numParticlesLocal: "
                        << numParticlesLocal << "  numNodes:" << numNodes;

    timeKernels = p.timeKernels; //true;

    // allocate host data
    h_min_x = new float;
    h_max_x = new float;
    h_min_y = new float;
    h_max_y = new float;
    h_min_y = new float;
    h_max_y = new float;

    h_mass = new float[numNodes];

    h_domainListIndices = new unsigned long[DOMAIN_LIST_SIZE];
    h_domainListKeys = new unsigned long[DOMAIN_LIST_SIZE];
    h_domainListLevels = new int[DOMAIN_LIST_SIZE];
    h_domainListIndex = new int;
    for (int i=0; i<DOMAIN_LIST_SIZE; i++) {
        h_domainListIndices[i] = KEY_MAX;
        h_domainListKeys[i] = KEY_MAX;
        h_domainListLevels[i] = -1;
    }

    h_x = new float[numNodes];
    h_y = new float[numNodes];
    h_z = new float[numNodes];

    h_vx = new float[numNodes];
    h_vy = new float[numNodes];
    h_vz = new float[numNodes];

    h_ax = new float[numNodes];
    h_ay = new float[numNodes];
    h_az = new float[numNodes];

    h_child = new int[8*numNodes];
    
    h_start = new int[numNodes];
    h_sorted = new int[numNodes];
    h_count = new int[numNodes];
    //h_output = new float[2*numNodes];

    time_resetArrays = new float[parameters.iterations];
    time_computeBoundingBox = new float[parameters.iterations];
    time_buildTree = new float[parameters.iterations];
    time_centreOfMass = new float[parameters.iterations];
    time_sort = new float[parameters.iterations];
    time_computeForces = new float[parameters.iterations];
    time_update = new float[parameters.iterations];
    time_copyDeviceToHost = new float[parameters.iterations];
    time_all = new float [parameters.iterations];

    printf("rank: %i  numProcesses: %i\n", h_subDomainHandler->rank, h_subDomainHandler->numProcesses);

    h_procCounter = new int[h_subDomainHandler->numProcesses];

    // allocate device data
    gpuErrorcheck(hipMalloc((void**)&d_min_x, sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_max_x, sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_min_y, sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_max_y, sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_min_z, sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_max_z, sizeof(float)));

    gpuErrorcheck(hipMemset(d_min_x, 0, sizeof(float)));
    gpuErrorcheck(hipMemset(d_max_x, 0, sizeof(float)));
    gpuErrorcheck(hipMemset(d_min_y, 0, sizeof(float)));
    gpuErrorcheck(hipMemset(d_max_y, 0, sizeof(float)));
    gpuErrorcheck(hipMemset(d_min_z, 0, sizeof(float)));
    gpuErrorcheck(hipMemset(d_max_z, 0, sizeof(float)));

    gpuErrorcheck(hipMalloc((void**)&d_mass, numNodes*sizeof(float)));

    gpuErrorcheck(hipMalloc((void**)&d_domainListIndices, DOMAIN_LIST_SIZE*sizeof(unsigned long)));
    gpuErrorcheck(hipMalloc((void**)&d_domainListKeys, DOMAIN_LIST_SIZE*sizeof(unsigned long)));
    gpuErrorcheck(hipMalloc((void**)&d_domainListLevels, DOMAIN_LIST_SIZE*sizeof(int)));
    gpuErrorcheck(hipMalloc((void**)&d_domainListIndex, sizeof(int)));

    gpuErrorcheck(hipMalloc((void**)&d_tempArray, 2*numParticles*sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_sortArray, numParticles*sizeof(int)));
    gpuErrorcheck(hipMalloc((void**)&d_sortArrayOut, numParticles*sizeof(int)));

    gpuErrorcheck(hipMalloc((void**)&d_procCounter, h_subDomainHandler->numProcesses*sizeof(int)));
    gpuErrorcheck(hipMalloc((void**)&d_procCounterTemp, h_subDomainHandler->numProcesses*sizeof(int)));

    gpuErrorcheck(hipMalloc((void**)&d_x, numNodes*sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_y, numNodes*sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_z, numNodes*sizeof(float)));

    gpuErrorcheck(hipMalloc((void**)&d_vx, numNodes*sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_vy, numNodes*sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_vz, numNodes*sizeof(float)));

    gpuErrorcheck(hipMalloc((void**)&d_ax, numNodes*sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_ay, numNodes*sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_az, numNodes*sizeof(float)));

    gpuErrorcheck(hipMalloc((void**)&d_index, sizeof(int)));
    gpuErrorcheck(hipMalloc((void**)&d_child, 8*numNodes*sizeof(int)));
    gpuErrorcheck(hipMalloc((void**)&d_start, numNodes*sizeof(int)));
    gpuErrorcheck(hipMalloc((void**)&d_sorted, numNodes*sizeof(int)));
    gpuErrorcheck(hipMalloc((void**)&d_count, numNodes*sizeof(int)));
    gpuErrorcheck(hipMalloc((void**)&d_mutex, sizeof(int)));

    //gpuErrorcheck(hipMalloc((void**)&d_subDomainHandler, sizeof(SubDomainKeyTree)));
    gpuErrorcheck(hipMalloc((void**)&d_subDomainHandler, sizeof(SubDomainKeyTree)));
    int size = 2 * sizeof(int) + 3 * sizeof(unsigned long);
    gpuErrorcheck(hipMalloc((void**)&d_range, size));
    //gpuErrorcheck(hipMemset(d_subDomainHandler->rank, 0, sizeof(int)));
    //gpuErrorcheck(hipMemset(d_subDomainHandler->range, {0, KEY_MAX/2, KEY_MAX}, 3*sizeof(unsigned long)));
    //gpuErrorcheck(hipMemset(d_subDomainHandler->numProcesses, 2, sizeof(int)));


    gpuErrorcheck(hipMemset(d_start, -1, numNodes*sizeof(int)));
    gpuErrorcheck(hipMemset(d_sorted, 0, numNodes*sizeof(int)));

    int memSize = sizeof(float) * 2 * numParticles;

    //gpuErrorcheck(hipMalloc((void**)&d_output, 2*numNodes*sizeof(float)));

    //plummerModel(h_mass, h_x, h_y, h_z, h_vx, h_vy, h_vz, h_ax, h_ay, h_az, numParticles);
    diskModel(h_mass, h_x, h_y, h_z, h_vx, h_vy, h_vz, h_ax, h_ay, h_az, numParticlesLocal); //numParticles);


    // copy data to GPU device

    hipMemcpy(d_subDomainHandler, h_subDomainHandler, sizeof(SubDomainKeyTree), hipMemcpyHostToDevice);
    hipMemcpy(d_range, h_subDomainHandler->range, size, hipMemcpyHostToDevice);
    hipMemcpy(&(d_subDomainHandler->range), &d_range, sizeof(unsigned long*), hipMemcpyHostToDevice);

    //hipMemcpy(d_subDomainHandler, h_subDomainHandler, sizeof(*h_subDomainHandler), hipMemcpyHostToDevice);
    //hipMemcpy(&d_subDomainHandler->rank, &h_subDomainHandler->rank, sizeof(int), hipMemcpyHostToDevice);
    //hipMemcpy(&d_subDomainHandler->numProcesses, &h_subDomainHandler->numProcesses, sizeof(int), hipMemcpyHostToDevice);
    //hipMemcpy(&d_subDomainHandler->range, &h_subDomainHandler->range, 3*sizeof(unsigned long), hipMemcpyHostToDevice);

    //hipMemcpy(d_mass, h_mass, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_mass, h_mass, numParticlesLocal*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_domainListIndices, h_domainListIndices, DOMAIN_LIST_SIZE*sizeof(unsigned long), hipMemcpyHostToDevice);
    hipMemcpy(d_domainListKeys, h_domainListKeys, DOMAIN_LIST_SIZE*sizeof(unsigned long), hipMemcpyHostToDevice);
    hipMemcpy(d_domainListLevels, h_domainListLevels, DOMAIN_LIST_SIZE*sizeof(int), hipMemcpyHostToDevice);
    gpuErrorcheck(hipMemset(d_domainListIndex, 0, sizeof(int)));

    /*hipMemcpy(d_x, h_x, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_z, h_z, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vx, h_vx, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vy, h_vy, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vz, h_vz, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_ax, h_ax, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_ay, h_ay, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_az, h_az, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);*/

    hipMemcpy(d_x,  h_x,  numParticlesLocal*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y,  h_y,  numParticlesLocal*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_z,  h_z,  numParticlesLocal*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vx, h_vx, numParticlesLocal*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vy, h_vy, numParticlesLocal*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vz, h_vz, numParticlesLocal*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_ax, h_ax, numParticlesLocal*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_ay, h_ay, numParticlesLocal*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_az, h_az, numParticlesLocal*sizeof(float), hipMemcpyHostToDevice);

}

BarnesHut::~BarnesHut() {
    delete h_min_x;
    delete h_max_x;
    delete h_min_y;
    delete h_max_y;
    delete h_min_z;
    delete h_max_z;

    delete [] h_subDomainHandler->range;
    delete h_subDomainHandler;

    delete [] h_mass;

    delete [] h_x;
    delete [] h_y;
    delete [] h_z;

    delete [] h_vx;
    delete [] h_vy;
    delete [] h_vz;

    delete [] h_ax;
    delete [] h_ay;
    delete [] h_az;

    delete [] h_child;
    delete [] h_start;
    delete [] h_sorted;
    delete [] h_count;
    //delete [] h_output;

    delete [] time_resetArrays;
    delete [] time_computeBoundingBox;
    delete [] time_buildTree;
    delete [] time_centreOfMass;
    delete [] time_sort;
    delete [] time_computeForces;
    delete [] time_update;
    delete [] time_copyDeviceToHost;
    delete [] time_all;

    gpuErrorcheck(hipFree(d_min_x));
    gpuErrorcheck(hipFree(d_max_x));
    gpuErrorcheck(hipFree(d_min_y));
    gpuErrorcheck(hipFree(d_max_y));
    gpuErrorcheck(hipFree(d_min_z));
    gpuErrorcheck(hipFree(d_max_z));

    gpuErrorcheck(hipFree(d_mass));

    gpuErrorcheck(hipFree(d_subDomainHandler->range));
    gpuErrorcheck(hipFree(d_subDomainHandler));
    gpuErrorcheck(hipFree(d_range));

    gpuErrorcheck(hipFree(d_x));
    gpuErrorcheck(hipFree(d_y));
    gpuErrorcheck(hipFree(d_z));

    gpuErrorcheck(hipFree(d_vx));
    gpuErrorcheck(hipFree(d_vy));
    gpuErrorcheck(hipFree(d_vz));

    gpuErrorcheck(hipFree(d_ax));
    gpuErrorcheck(hipFree(d_ay));
    gpuErrorcheck(hipFree(d_az));

    gpuErrorcheck(hipFree(d_index));
    gpuErrorcheck(hipFree(d_child));
    gpuErrorcheck(hipFree(d_start));
    gpuErrorcheck(hipFree(d_sorted));
    gpuErrorcheck(hipFree(d_count));

    gpuErrorcheck(hipFree(d_mutex));

    //gpuErrorcheck(hipFree(d_output));

    hipDeviceSynchronize();
}

void BarnesHut::update(int step)
{

    int device;
    hipGetDevice(&device);
    Logger(INFO) << "&d_sortArrayOut = " << d_sortArrayOut << " on device: " << device;

    /*RESETTING ARRAYS*************************************************************************/
    float elapsedTime;
    hipEventCreate(&start_global);
    hipEventCreate(&stop_global);
    hipEventRecord(start_global, 0);

    float elapsedTimeKernel;

    elapsedTimeKernel = KernelHandler.resetArrays(d_mutex, d_x, d_y, d_z, d_mass, d_count, d_start, d_sorted, d_child, d_index,
                        d_min_x, d_max_x, d_min_y, d_max_y, d_min_z, d_max_z, numParticles, numNodes,
                        d_procCounter, d_procCounterTemp,timeKernels);

    KernelHandler.resetArraysParallel(d_domainListIndex, d_domainListKeys, d_domainListIndices,
                                      d_domainListLevels, d_tempArray, numParticles, numNodes);

    time_resetArrays[step] = elapsedTimeKernel;
    if (timeKernels) {
        Logger(TIME) << "\tReset arrays: " << elapsedTimeKernel << " ms";
    }
    /*resetting arrays*************************************************************************/

    /*COMPUTE BOUNDING BOX*********************************************************************/
    elapsedTimeKernel = KernelHandler.computeBoundingBox(d_mutex, d_x, d_y, d_z, d_min_x, d_max_x, d_min_y, d_max_y,
                               d_min_z, d_max_z, numParticles, timeKernels);

    globalizeBoundingBox();

    time_computeBoundingBox[step] = elapsedTimeKernel;
    if (timeKernels) {
        Logger(TIME) << "\tBounding box: " << elapsedTimeKernel << " ms";
    }
    /*compute bounding box*********************************************************************/

    /*COMPUTE BOUNDING BOX*********************************************************************/

    KernelHandler.particlesPerProcess(d_x, d_y, d_z, d_mass, d_count, d_start, d_child, d_index, d_min_x, d_max_x, d_min_y, d_max_y,
                                      d_min_z, d_max_z, numParticlesLocal, numNodes, d_subDomainHandler, d_procCounter, d_procCounterTemp);

    KernelHandler.sortParticlesProc(d_x, d_y, d_z, d_mass, d_count, d_start, d_child, d_index, d_min_x, d_max_x, d_min_y, d_max_y,
                                      d_min_z, d_max_z, numParticlesLocal, numNodes, d_subDomainHandler, d_procCounter, d_procCounterTemp,
                                      d_sortArray);

    //KernelHandler.sendParticles(d_x, d_y, d_z, d_mass, d_count, d_start, d_child, d_index, d_min_x, d_max_x, d_min_y, d_max_y,
    //                            d_min_z, d_max_z, numParticles, numNodes, d_subDomainHandler, d_procCounter, d_tempArray,
    //                            d_sortArray, d_sortArrayOut);

    //KernelHandler.treeInfo(d_x, d_y, d_z, d_mass, d_count, d_start, d_child, d_index, d_min_x, d_max_x, d_min_y, d_max_y,
      //                     d_min_z, d_max_z, numParticlesLocal, numNodes, d_procCounter, d_subDomainHandler);

//#if TESTING
    Logger(INFO) << "RADIX SORT";

    float elapsedTimeSorting = 0.f;
    hipEvent_t start_t_sorting, stop_t_sorting; // used for timing
    hipEventCreate(&start_t_sorting);
    hipEventCreate(&stop_t_sorting);
    hipEventRecord(start_t_sorting, 0);

    // position
    sortArrayRadix(d_x, d_tempArray, d_sortArray, d_sortArrayOut, numParticlesLocal);
    KernelHandler.copyArray(d_x, d_tempArray, numParticlesLocal);
    sortArrayRadix(d_y, d_tempArray, d_sortArray, d_sortArrayOut, numParticlesLocal);
    KernelHandler.copyArray(d_y, d_tempArray, numParticlesLocal);
    sortArrayRadix(d_z, d_tempArray, d_sortArray, d_sortArrayOut, numParticlesLocal);
    KernelHandler.copyArray(d_z, d_tempArray, numParticlesLocal);

    // velocity
    sortArrayRadix(d_vx, d_tempArray, d_sortArray, d_sortArrayOut, numParticlesLocal);
    KernelHandler.copyArray(d_vx, d_tempArray, numParticlesLocal);
    sortArrayRadix(d_vy, d_tempArray, d_sortArray, d_sortArrayOut, numParticlesLocal);
    KernelHandler.copyArray(d_vy, d_tempArray, numParticlesLocal);
    sortArrayRadix(d_vz, d_tempArray, d_sortArray, d_sortArrayOut, numParticlesLocal);
    KernelHandler.copyArray(d_vz, d_tempArray, numParticlesLocal);

    // acceleration
    sortArrayRadix(d_ax, d_tempArray, d_sortArray, d_sortArrayOut, numParticlesLocal);
    KernelHandler.copyArray(d_ax, d_tempArray, numParticlesLocal);
    sortArrayRadix(d_ay, d_tempArray, d_sortArray, d_sortArrayOut, numParticlesLocal);
    KernelHandler.copyArray(d_ay, d_tempArray, numParticlesLocal);
    sortArrayRadix(d_az, d_tempArray, d_sortArray, d_sortArrayOut, numParticlesLocal);
    KernelHandler.copyArray(d_az, d_tempArray, numParticlesLocal);

    // mass
    sortArrayRadix(d_mass, d_tempArray, d_sortArray, d_sortArrayOut, numParticlesLocal);
    KernelHandler.copyArray(d_mass, d_tempArray, numParticlesLocal);

    hipEventRecord(stop_t_sorting, 0);
    hipEventSynchronize(stop_t_sorting);
    hipEventElapsedTime(&elapsedTimeSorting, start_t_sorting, stop_t_sorting);
    hipEventDestroy(start_t_sorting);
    hipEventDestroy(stop_t_sorting);

    Logger(TIME) << "\tSorting for process: " << elapsedTimeSorting << "ms";

    //hipMemcpy(h_x, d_x, 2*numParticles*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_procCounter, d_procCounter, h_subDomainHandler->numProcesses*sizeof(int), hipMemcpyDeviceToHost);

    for (int proc=0; proc<h_subDomainHandler->numProcesses; proc++) {
        printf("[rank %i] HOST: procCounter[%i] = %i\n", h_subDomainHandler->rank, proc, h_procCounter[proc]);
    }


    float elapsedTimeSending = 0.f;
    hipEvent_t start_t_sending, stop_t_sending; // used for timing
    hipEventCreate(&start_t_sending);
    hipEventCreate(&stop_t_sending);
    hipEventRecord(start_t_sending, 0);


    //send particles
    /*------------------------------------------------------------------------------------------------------------*/
    int *sendLengths;
    sendLengths = new int[h_subDomainHandler->numProcesses];
    sendLengths[h_subDomainHandler->rank] = 0;
    int *receiveLengths;
    receiveLengths = new int[h_subDomainHandler->numProcesses];
    receiveLengths[h_subDomainHandler->rank] = 0;

    for (int proc=0; proc < h_subDomainHandler->numProcesses; proc++) {
        if (proc != h_subDomainHandler->rank) {
            sendLengths[proc] = h_procCounter[proc];
        }
    }

    int reqCounter = 0;
    MPI_Request reqMessageLengths[h_subDomainHandler->numProcesses-1];
    MPI_Status statMessageLengths[h_subDomainHandler->numProcesses-1];

    //send plistLengthSend and receive plistLengthReceive
    for (int proc=0; proc < h_subDomainHandler->numProcesses; proc++) {
        if (proc != h_subDomainHandler->rank) {
            MPI_Isend(&sendLengths[proc], 1, MPI_INT, proc, 17, MPI_COMM_WORLD, &reqMessageLengths[reqCounter]);
            MPI_Recv(&receiveLengths[proc], 1, MPI_INT, proc, 17, MPI_COMM_WORLD, &statMessageLengths[reqCounter]);
            reqCounter++;
        }
    }
    MPI_Waitall(h_subDomainHandler->numProcesses-1, reqMessageLengths, statMessageLengths);

    for (int proc=0; proc < h_subDomainHandler->numProcesses; proc++) {
        printf("[rank %i] reveiceLengths[%i] = %i  sendLengths[%i] = %i\n", h_subDomainHandler->rank,
               proc, receiveLengths[proc], proc, sendLengths[proc]);
    }

    KernelHandler.treeInfo(d_x, d_y, d_z, d_mass, d_count, d_start, d_child, d_index, d_min_x, d_max_x, d_min_y, d_max_y,
                           d_min_z, d_max_z, numParticlesLocal, numNodes, d_procCounter, d_subDomainHandler, d_sortArray,
                           d_sortArrayOut);


    /*-------------------------------------------------------*/

    //send and receive particles
    //d_x;
    //d_tempArray;

#if CUDA_AWARE_MPI_TESTING
    // ------------------CUDA aware MPI Testing ----------------------------------------------------------------------
    MPI_Request reqTest[h_subDomainHandler->numProcesses - 1];
    MPI_Status statTest[h_subDomainHandler->numProcesses - 1];

    reqCounter = 0;

    for (int proc=0; proc < h_subDomainHandler->numProcesses; proc++) {
        if (proc != h_subDomainHandler->rank) {
            MPI_Isend(d_sortArrayOut, 10, MPI_INT, proc, 0, MPI_COMM_WORLD, &reqTest[reqCounter]);
            MPI_Recv(d_sortArray, 10, MPI_INT, proc, 0, MPI_COMM_WORLD, &statTest[reqCounter]);
            reqCounter++;
        }
    }

    MPI_Waitall(h_subDomainHandler->numProcesses-1, reqTest, statTest);
    // ------------------CUDA aware MPI Testing ----------------------------------------------------------------------
#endif

    Logger(INFO) << "&d_sortArrayOut = " << d_sortArrayOut;

    /*MPI_Request reqParticles[h_subDomainHandler->numProcesses - 1];
    MPI_Status statParticles[h_subDomainHandler->numProcesses - 1];

    reqCounter = 0;
    //int sendOffset = 0;
    int receiveOffset = 0;

    for (int proc=0; proc < h_subDomainHandler->numProcesses; proc++) {
        if (proc != h_subDomainHandler->rank) {
            if (proc == 0) {
                MPI_Isend(&d_x[0], sendLengths[proc], MPI_FLOAT, proc, 17,
                          MPI_COMM_WORLD, &reqParticles[reqCounter]);
            }
            else {
                MPI_Isend(&d_x[h_procCounter[proc-1]], sendLengths[proc], MPI_FLOAT, proc, 17,
                          MPI_COMM_WORLD, &reqParticles[reqCounter]);
            }
            MPI_Recv(&d_tempArray[0] + receiveOffset, receiveLengths[proc], MPI_FLOAT, proc, 17,
                     MPI_COMM_WORLD, &statParticles[reqCounter]);
            receiveOffset += receiveLengths[proc];
            reqCounter++;
        }
    }

    MPI_Waitall(h_subDomainHandler->numProcesses-1, reqParticles, statParticles);*/

    sendParticlesEntry(sendLengths, receiveLengths, d_x);
    sendParticlesEntry(sendLengths, receiveLengths, d_y);
    sendParticlesEntry(sendLengths, receiveLengths, d_z);

    sendParticlesEntry(sendLengths, receiveLengths, d_vx);
    sendParticlesEntry(sendLengths, receiveLengths, d_vz);
    sendParticlesEntry(sendLengths, receiveLengths, d_vy);

    sendParticlesEntry(sendLengths, receiveLengths, d_ax);
    sendParticlesEntry(sendLengths, receiveLengths, d_ay);
    sendParticlesEntry(sendLengths, receiveLengths, d_az);

    numParticlesLocal = sendParticlesEntry(sendLengths, receiveLengths, d_mass);



//#if TESTING

    // delete sent entries and copy received entries
    /*if (h_subDomainHandler->rank != 0) {
        KernelHandler.copyArray(d_x, &d_x[h_procCounter[h_subDomainHandler->rank - 1]], h_procCounter[h_subDomainHandler->rank]); //float *targetArray, float *sourceArray, int n)
    }

    KernelHandler.resetFloatArray(d_x, 0, h_procCounter[h_subDomainHandler->rank]); //resetFloatArrayKernel(float *array, float value, int n)
    KernelHandler.copyArray(&d_x[h_procCounter[h_subDomainHandler->rank]], d_tempArray, receiveOffset);

    printf("FINISHED!!!\n");
    */
    delete[] sendLengths;
    delete[] receiveLengths;
    /*------------------------------------------------------------------------------------------------------------*/

    hipEventRecord(stop_t_sending, 0);
    hipEventSynchronize(stop_t_sending);
    hipEventElapsedTime(&elapsedTimeSending, start_t_sending, stop_t_sending);
    hipEventDestroy(start_t_sending);
    hipEventDestroy(stop_t_sending);

    Logger(TIME) << "\tSending particles: " << elapsedTimeSending <<  "ms";

    //KernelHandler.sendParticles(d_x, d_y, d_z, d_mass, d_count, d_start, d_child, d_index, d_min_x, d_max_x, d_min_y, d_max_y,
      //                          d_min_z, d_max_z, numParticles, numNodes, d_subDomainHandler, d_procCounter, d_tempArray, d_sortArray, d_sortArrayOut);


//#endif

    KernelHandler.sendParticles(d_x, d_y, d_z, d_mass, d_count, d_start, d_child, d_index, d_min_x, d_max_x, d_min_y, d_max_y,
                                d_min_z, d_max_z, numParticlesLocal, numNodes, d_subDomainHandler, d_procCounter, d_tempArray,
                                d_sortArray, d_sortArrayOut);

    KernelHandler.treeInfo(d_x, d_y, d_z, d_mass, d_count, d_start, d_child, d_index, d_min_x, d_max_x, d_min_y, d_max_y,
                           d_min_z, d_max_z, numParticlesLocal, numNodes, d_procCounter, d_subDomainHandler, d_sortArray,
                           d_sortArrayOut);

    /*BUILDING TREE*************************************************************************/
    KernelHandler.createDomainList(d_subDomainHandler, 21, d_domainListKeys, d_domainListLevels, d_domainListIndex);
    //KernelHandler.buildDomainTree(d_domainListIndex, d_domainListKeys, d_domainListLevels, d_count, d_start, d_child,
      //                            d_index, numParticles, numNodes);

    elapsedTimeKernel = KernelHandler.buildTree(d_x, d_y, d_z, d_mass, d_count, d_start, d_child, d_index, d_min_x, d_max_x, d_min_y, d_max_y,
                      d_min_z, d_max_z, numParticlesLocal, numParticles, timeKernels); //numParticles -> numParticlesLocal

    KernelHandler.buildDomainTree(d_domainListIndex, d_domainListKeys, d_domainListLevels, d_count, d_start, d_child,
                                d_index, numParticlesLocal, numNodes); //TODO: numParticlesLocal or numParticles?

    //KernelHandler.buildDomainTree(d_domainListIndex, d_domainListKeys, d_domainListLevels, d_count, d_start, d_child,
      //                            d_index, numParticles, numNodes);

    //KernelHandler.treeInfo(d_x, d_y, d_z, d_mass, d_count, d_start, d_child, d_index, d_min_x, d_max_x, d_min_y, d_max_y,
      //                     d_min_z, d_max_z, numParticlesLocal, numNodes, d_procCounter, d_subDomainHandler, d_sortArray,
      //                     d_sortArrayOut);

    //KernelHandler.getParticleKey(d_x, d_y, d_z, d_min_x, d_max_x, d_min_y, d_max_y,
                         //      d_min_z, d_max_z, 0UL, 21, numParticles, d_subDomainHandler);


    //KernelHandler.traverseIterative(d_x, d_y, d_z, d_mass, d_child, numParticles, numNodes, d_subDomainHandler, 21);
    //KernelHandler.createDomainList(d_x, d_y, d_z, d_mass, d_child, numParticles, d_subDomainHandler, 21);

    //KernelHandler.createDomainList(d_x, d_y, d_z, d_mass, d_min_x, d_max_x,
      //                                  d_min_y, d_max_y, d_min_z, d_max_z, d_child, numParticles,
        //                                d_subDomainHandler, 21);

    //KernelHandler.createDomainList(d_subDomainHandler, 21, d_domainListIndices, d_domainListLevels, d_domainListIndex);


    time_buildTree[step] = elapsedTimeKernel;
    if (timeKernels) {
        Logger(TIME) << "\tBuilding tree: " << elapsedTimeKernel << " ms";
    }
    /*building tree*************************************************************************/

    /*CENTER OF MASS************************************************************************/
    elapsedTimeKernel = KernelHandler.centreOfMass(d_x, d_y, d_z, d_mass, d_index, numParticles, timeKernels);

    time_centreOfMass[step] = elapsedTimeKernel;
    if (timeKernels) {
        Logger(TIME) << "\tCenter of mass: " << elapsedTimeKernel << " ms";
    }
    /*center of mass************************************************************************/

    /*SORTING*******************************************************************************/
    elapsedTimeKernel = KernelHandler.sort(d_count, d_start, d_sorted, d_child, d_index, numParticles, timeKernels);
    //elapsedTimeKernel = 0;

    time_sort[step] = elapsedTimeKernel;
    if (timeKernels) {
        Logger(TIME) << "\tSort particles: " << elapsedTimeKernel << " ms";
    }
    /*sorting*******************************************************************************/

    /*COMPUTING FORCES**********************************************************************/
    elapsedTimeKernel = KernelHandler.computeForces(d_x, d_y, d_z, d_vx, d_vy, d_vz, d_ax, d_ay, d_az, d_mass, d_sorted, d_child,
                          d_min_x, d_max_x, numParticlesLocal, parameters.gravity, timeKernels); //TODO: numParticlesLocal or numParticles?

    time_computeForces[step] = elapsedTimeKernel;
    if (timeKernels) {
        Logger(TIME) << "\tCompute forces: " << elapsedTimeKernel << " ms";
    }
    /*computing forces**********************************************************************/

    /*UPDATING******************************************************************************/
    elapsedTimeKernel = KernelHandler.update(d_x, d_y, d_z, d_vx, d_vy, d_vz, d_ax, d_ay, d_az, numParticlesLocal,
                   parameters.timestep, parameters.dampening, timeKernels); //TODO: numParticlesLocal or numParticles?


    time_update[step] = elapsedTimeKernel;
    if (timeKernels) {
        Logger(TIME) << "\tUpdate particles: " << elapsedTimeKernel << " ms";
    }
    /*updating******************************************************************************/

    /*COPYING TO HOST***********************************************************************/
    hipEvent_t start_t, stop_t; // used for timing
    hipEventCreate(&start_t);
    hipEventCreate(&stop_t);
    hipEventRecord(start_t, 0);

    hipMemcpy(h_x, d_x, numNodes*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_y, d_y, numNodes*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_z, d_z, numNodes*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_vx, d_vx, numNodes*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_vy, d_vy, numNodes*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_vz, d_vz, numNodes*sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(stop_t, 0);
    hipEventSynchronize(stop_t);
    hipEventElapsedTime(&elapsedTimeKernel, start_t, stop_t);
    hipEventDestroy(start_t);
    hipEventDestroy(stop_t);
    /*copying to host***********************************************************************/

    hipDeviceSynchronize();

    time_copyDeviceToHost[step] = elapsedTimeKernel;
    if (timeKernels) {
        Logger(TIME) << "\tCopying to host: " << elapsedTimeKernel << " ms";
    }

    //std::cout << "x[0]: " << h_x[0] << std::endl;
    //std::cout << "v[0]: " << h_vx[0] << std::endl;


    hipEventRecord(stop_global, 0);
    hipEventSynchronize(stop_global);
    hipEventElapsedTime(&elapsedTime, start_global, stop_global);
    hipEventDestroy(start_global);
    hipEventDestroy(stop_global);

    time_all[step] = elapsedTime;
    Logger(TIME) << "Elapsed time for step " << step << " : " << elapsedTime << " ms";
    Logger(INFO) << "-----------------------------------------------------------------------------------------";

    float *xAll, *yAll, *zAll;
    gatherParticles(xAll, yAll, zAll);

    step++;
}


void BarnesHut::plummerModel(float *mass, float *x, float* y, float *z,
                                    float *x_vel, float *y_vel, float *z_vel,
                                    float *x_acc, float *y_acc, float *z_acc, int n)
{
    float a = 1.0;
    float pi = 3.14159265;
    std::default_random_engine generator;
    std::uniform_real_distribution<float> distribution(0, 1.0);
    std::uniform_real_distribution<float> distribution2(0, 0.1);
    std::uniform_real_distribution<float> distribution_phi(0.0, 2 * pi);
    std::uniform_real_distribution<float> distribution_theta(-1.0, 1.0);

    // loop through all particles
    for (int i = 0; i < n; i++){
        float phi = distribution_phi(generator);
        float theta = acos(distribution_theta(generator));
        float r = a / sqrt(pow(distribution(generator), -0.666666) - 1);

        // set mass and position of particle
        mass[i] = 1.0;
        x[i] = r*cos(phi);
        y[i] = r*sin(phi);
        if (i%2==0) {
            z[i] = i*0.001;
        }
        else {
            z[i] = i*-0.001;
        }

        // set velocity of particle
        float s = 0.0;
        float t = 0.1;
        while(t > s*s*pow(1.0 - s*s, 3.5)){
            s = distribution(generator);
            t = distribution2(generator);
        }
        float v = 100*s*sqrt(2)*pow(1.0 + r*r, -0.25);
        phi = distribution_phi(generator);
        theta = acos(distribution_theta(generator));
        x_vel[i] = v*cos(phi);
        y_vel[i] = v*sin(phi);
        z_vel[i] = 0.0;

        // set acceleration to zero
        x_acc[i] = 0.0;
        y_acc[i] = 0.0;
        z_acc[i] = 0.0;

    }

}

void BarnesHut::diskModel(float *mass, float *x, float* y, float* z, float *x_vel, float *y_vel, float *z_vel,
                                 float *x_acc, float *y_acc, float *z_acc, int n)
{
    float a = 1.0;
    float pi = 3.14159265;
    std::default_random_engine generator;
    std::uniform_real_distribution<float> distribution(1.5, 12.0);
    std::uniform_real_distribution<float> distribution_theta(0.0, 2 * pi);

    float solarMass = 100000;

    // loop through all particles
    for (int i = 0; i < n; i++) {

        float theta = distribution_theta(generator);
        float r = distribution(generator);

        // set mass and position of particle
        if (h_subDomainHandler->rank == 0) {
            if (i == 0) {
                mass[i] = solarMass; //100000;
                x[i] = 0;
                y[i] = 0;
                z[i] = 0;
            } else {
                mass[i] = 2 * solarMass / numParticles;
                x[i] = r * cos(theta);
                y[i] = r * sin(theta);

                if (i % 2 == 0) {
                    z[i] = i * 1e-7;
                } else {
                    z[i] = i * -1e-7;
                }
            }
        }
        else {
            mass[i] = 2 * solarMass / numParticles;
            x[i] = (r + h_subDomainHandler->rank * 1.1e-1) * cos(theta);
            y[i] = (r + h_subDomainHandler->rank * 1.3e-1) * sin(theta);

            if (i % 2 == 0) {
                z[i] = i * 1e-7 * h_subDomainHandler->rank;
            } else {
                z[i] = i * -1e-7 * h_subDomainHandler->rank;
            }
        }


        // set velocity of particle
        float rotation = 1;  // 1: clockwise   -1: counter-clockwise
        float v = sqrt(solarMass / (r));

        if (i == 0) {
            x_vel[0] = 0.0;
            y_vel[0] = 0.0;
            z_vel[0] = 0.0;
        }
        else{
            x_vel[i] = rotation*v*sin(theta);
            y_vel[i] = -rotation*v*cos(theta);
            z_vel[i] = 0.0;
        }

        // set acceleration to zero
        x_acc[i] = 0.0;
        y_acc[i] = 0.0;
        z_acc[i] = 0.0;
    }

}

float BarnesHut::getSystemSize() {

    float x_max = 0;
    float y_max = 0;
    float z_max = 0;

    for (int i = 0; i < numParticles; i++) {
        if (abs(h_x[i]) > x_max) {
            x_max = abs(h_x[i]);
        }
        if (abs(h_y[i]) > y_max) {
            y_max = abs(h_y[i]);
        }
        if (abs(h_z[i]) > z_max) {
            z_max = abs(h_z[i]);
        }
    }

    float systemSize = x_max;
    if (y_max > systemSize) {
        systemSize = y_max;
    }
    if (z_max > systemSize) {
        systemSize = z_max;
    }

    float globalSystemSize;
    MPI_Allreduce(&systemSize, &globalSystemSize, 1, MPI_FLOAT, MPI_MAX, MPI_COMM_WORLD);

    //return systemSize;
    return globalSystemSize;

}

void BarnesHut::globalizeBoundingBox() {
    // X MIN
    MPI_Allreduce(MPI_IN_PLACE, d_min_x, 1, MPI_FLOAT, MPI_MIN, MPI_COMM_WORLD);
    // X MAX
    MPI_Allreduce(MPI_IN_PLACE, d_max_x, 1, MPI_FLOAT, MPI_MAX, MPI_COMM_WORLD);
    // Y MIN
    MPI_Allreduce(MPI_IN_PLACE, d_min_y, 1, MPI_FLOAT, MPI_MIN, MPI_COMM_WORLD);
    // Y MAX
    MPI_Allreduce(MPI_IN_PLACE, d_max_y, 1, MPI_FLOAT, MPI_MAX, MPI_COMM_WORLD);
    // Z MIN
    MPI_Allreduce(MPI_IN_PLACE, d_min_z, 1, MPI_FLOAT, MPI_MIN, MPI_COMM_WORLD);
    // Z MAX
    MPI_Allreduce(MPI_IN_PLACE, d_max_z, 1, MPI_FLOAT, MPI_MAX, MPI_COMM_WORLD);
}

void BarnesHut::sortArrayRadix(float *arrayToSort, float *tempArray, int *keyIn, int *keyOut, int n) {
    // Determine temporary device storage requirements
    void     *d_temp_storage = NULL;
    size_t   temp_storage_bytes = 0;
    gpuErrorcheck(hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
                                    keyIn, keyOut, arrayToSort, tempArray, n));
    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    gpuErrorcheck(hipMalloc((void**)&d_temp_storage, temp_storage_bytes));

    // Run sorting operation
    gpuErrorcheck(hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
                                    keyIn, keyOut, arrayToSort, tempArray, n));

    hipFree(d_temp_storage);
}
//alternatively using Thrust:
/*thrust::device_vector<int>  indices(N);
thrust::sequence(indices.begin(),indices.end());
thrust::sort_by_key(keys.begin(),keys.end(),indices.begin());

thrust::device_vector<int> temp(N);
thrust::device_vector<int> *sorted = &temp;
thrust::device_vector<int> *pa_01 = &a_01;
thrust::device_vector<int> *pa_02 = &a_02;
...
thrust::device_vector<int> *pa_20 = &a_20;

thrust::gather(indices.begin(), indices.end(), *pa_01, *sorted);
pa_01 = sorted; sorted = &a_01;
thrust::gather(indices.begin(), indices.end(), *pa_02, *sorted);
pa_02 = sorted; sorted = &a_02;
...
thrust::gather(indices.begin(), indices.end(), *pa_20, *sorted);
pa_20 = sorted; sorted = &a_20;*/


void BarnesHut::gatherParticles(float *xAll, float *yAll, float *zAll) {

    //calculate amount of particles for own process

    //gather these information
    //MPI_Gather(&pLength, 1, MPI_INT, pArrayReceiveLength, 1, MPI_INT, 0, MPI_COMM_WORLD);

    //calculate total receive length and allocate memory

    //collect information
    //MPI_Gatherv(pArray, pLength, mpiParticle, pArrayAll, pArrayReceiveLength,
                //pArrayDisplacements, mpiParticle, 0, MPI_COMM_WORLD);
}

int BarnesHut::sendParticlesEntry(int *sendLengths, int *receiveLengths, float *entry) {
    MPI_Request reqParticles[h_subDomainHandler->numProcesses - 1];
    MPI_Status statParticles[h_subDomainHandler->numProcesses - 1];

    int reqCounter = 0;
    int receiveOffset = 0;

    for (int proc=0; proc < h_subDomainHandler->numProcesses; proc++) {
        if (proc != h_subDomainHandler->rank) {
            if (proc == 0) {
                MPI_Isend(&entry[0], sendLengths[proc], MPI_FLOAT, proc, 17,
                          MPI_COMM_WORLD, &reqParticles[reqCounter]);
            }
            else {
                MPI_Isend(&entry[h_procCounter[proc-1]], sendLengths[proc], MPI_FLOAT, proc, 17,
                          MPI_COMM_WORLD, &reqParticles[reqCounter]);
            }
            MPI_Recv(&d_tempArray[0] + receiveOffset, receiveLengths[proc], MPI_FLOAT, proc, 17,
                     MPI_COMM_WORLD, &statParticles[reqCounter]);
            receiveOffset += receiveLengths[proc];
            reqCounter++;
        }
    }

    MPI_Waitall(h_subDomainHandler->numProcesses-1, reqParticles, statParticles);

    int offset = 0;
    for (int i=0; i < h_subDomainHandler->rank; i++) {
        offset += h_procCounter[h_subDomainHandler->rank];
    }


    if (h_subDomainHandler->rank != 0) {
        KernelHandler.copyArray(&entry[0], &entry[offset - h_procCounter[h_subDomainHandler->rank]] /*&entry[h_procCounter[h_subDomainHandler->rank - 1]]*/, h_procCounter[h_subDomainHandler->rank]); //float *targetArray, float *sourceArray, int n)
    }

    KernelHandler.resetFloatArray(&entry[h_procCounter[h_subDomainHandler->rank]], 0, numParticles-h_procCounter[h_subDomainHandler->rank]); //resetFloatArrayKernel(float *array, float value, int n)
    KernelHandler.copyArray(&entry[h_procCounter[h_subDomainHandler->rank]], d_tempArray, receiveOffset);

    Logger(INFO) << "New local particle amount: " << receiveOffset + h_procCounter[h_subDomainHandler->rank]
                        << "  (receiveOffset = " << receiveOffset << ", procCounter = "
                        << h_procCounter[h_subDomainHandler->rank] << ")";

    return receiveOffset + h_procCounter[h_subDomainHandler->rank];
}




