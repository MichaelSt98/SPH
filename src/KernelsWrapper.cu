/**
 * Wrapping CUDA Kernel functions.
 */

#include "../include/KernelsWrapper.cuh"

/*
dim3 gridSize  = 1024; //2048; //1024; //512;
dim3 blockSize = 256; //256;
 */

KernelsWrapper::KernelsWrapper() {
    gridSize = 0;
    blockSize = 0;
    blockSizeInt = 0;
    warp = 0;
    stackSize = 0;
}

KernelsWrapper::KernelsWrapper(SimulationParameters p) {
    gridSize = p.gridSize;
    blockSize = p.blockSize;
    blockSizeInt = p.blockSize;
    warp = p.warp;
    stackSize = p.stackSize;
}

float KernelsWrapper::resetArrays(int *mutex, float *x, float *y, float *z, float *mass, int *count,
                          int *start, int *sorted, int *child, int *index, float *minX, float *maxX, float *minY, float *maxY,
                          float *minZ, float *maxZ, int n, int m, int *procCounter, int *procCounterTemp, bool timing) {

    float elapsedTime = 0.f;
    if (timing) {
        hipEvent_t start_t, stop_t; // used for timing
        hipEventCreate(&start_t);
        hipEventCreate(&stop_t);
        hipEventRecord(start_t, 0);

        resetArraysKernel<<< gridSize, blockSize >>>(mutex, x, y, z, mass, count, start, sorted, child, index,
                minX, maxX, minY, maxY, minZ, maxZ, n, m, procCounter, procCounterTemp);

        hipEventRecord(stop_t, 0);
        hipEventSynchronize(stop_t);
        hipEventElapsedTime(&elapsedTime, start_t, stop_t);
        hipEventDestroy(start_t);
        hipEventDestroy(stop_t);
    }
    else {
        resetArraysKernel<<< gridSize, blockSize >>>(mutex, x, y, z, mass, count, start, sorted, child, index,
                                                     minX, maxX, minY, maxY, minZ, maxZ, n, m, procCounter,
                                                     procCounterTemp);
    }
    return elapsedTime;

}

void KernelsWrapper::resetArraysParallel(int *domainListIndex, unsigned long *domainListKeys,
                                         int *domainListIndices, int *domainListLevels,
                                         float *tempArray, int *to_delete_cell, int *to_delete_leaf,
                                         int n, int m) {

    resetArraysParallelKernel<<< gridSize, blockSize >>>(domainListIndex, domainListKeys, domainListIndices,
                                                         domainListLevels, tempArray, to_delete_cell, to_delete_leaf,
                                                         n, m);
}

float KernelsWrapper::computeBoundingBox(int *mutex, float *x, float *y, float *z, float *minX,
                                 float *maxX, float *minY, float *maxY, float *minZ, float *maxZ, int n, bool timing) {

    float elapsedTime = 0.f;
    if (timing) {
        hipEvent_t start_t, stop_t; // used for timing
        hipEventCreate(&start_t);
        hipEventCreate(&stop_t);
        hipEventRecord(start_t, 0);

        computeBoundingBoxKernel<<< gridSize, blockSize, 6*sizeof(float)*blockSizeInt >>>(mutex, x, y, z, minX, maxX, minY, maxY, minZ, maxZ, n, blockSizeInt);

        hipEventRecord(stop_t, 0);
        hipEventSynchronize(stop_t);
        hipEventElapsedTime(&elapsedTime, start_t, stop_t);
        hipEventDestroy(start_t);
        hipEventDestroy(stop_t);
    }
    else {
        computeBoundingBoxKernel<<< gridSize, blockSize, 6*sizeof(float)*blockSizeInt >>>(mutex, x, y, z, minX, maxX, minY, maxY, minZ, maxZ, n, blockSizeInt);
    }
    return elapsedTime;

}

float KernelsWrapper::buildDomainTree(int *domainListIndex, unsigned long *domainListKeys, int *domainListLevels,
                                      int *domainListIndices, int *count, int *start, int *child, int *index, int n,
                                      int m, bool timing) {

    float elapsedTime = 0.f;
    if (timing) {
        hipEvent_t start_t, stop_t; // used for timing
        hipEventCreate(&start_t);
        hipEventCreate(&stop_t);
        hipEventRecord(start_t, 0);

    buildDomainTreeKernel<<< 1, 1 >>>(domainListIndex, domainListKeys, domainListLevels, domainListIndices, count, start, child, index, n, m);

        hipEventRecord(stop_t, 0);
        hipEventSynchronize(stop_t);
        hipEventElapsedTime(&elapsedTime, start_t, stop_t);
        hipEventDestroy(start_t);
        hipEventDestroy(stop_t);
    }
    else {
        buildDomainTreeKernel<<< 1, 1 >>>(domainListIndex, domainListKeys, domainListLevels, domainListIndices, count, start, child, index, n, m);
    }
    return elapsedTime;

}

float KernelsWrapper::treeInfo(float *x, float *y, float *z, float *mass, int *count, int *start,
                                int *child, int *index, float *minX, float *maxX, float *minY, float *maxY,
                                float *minZ, float *maxZ, int n, int m, int *procCounter, SubDomainKeyTree *s,
                                int *sortArray, int *sortArrayOut, bool timing) {

    float elapsedTime = 0.f;
    if (timing) {
        hipEvent_t start_t, stop_t; // used for timing
        hipEventCreate(&start_t);
        hipEventCreate(&stop_t);
        hipEventRecord(start_t, 0);

        treeInfoKernel<<< gridSize, blockSize >>>(x, y, z, mass, count, start, child, index,
                                        minX, maxX, minY, maxY, minZ, maxZ, n, m, procCounter, s, sortArray,
                                        sortArrayOut);

        hipEventRecord(stop_t, 0);
        hipEventSynchronize(stop_t);
        hipEventElapsedTime(&elapsedTime, start_t, stop_t);
        hipEventDestroy(start_t);
        hipEventDestroy(stop_t);
    }
    else {
        treeInfoKernel<<< gridSize, blockSize >>>(x, y, z, mass, count, start, child, index,
                                                  minX, maxX, minY, maxY, minZ, maxZ, n, m, procCounter, s, sortArray,
                                                  sortArrayOut);
    }
    return elapsedTime;

}

float KernelsWrapper::particlesPerProcess(float *x, float *y, float *z, float *mass, int *count, int *start,
                                   int *child, int *index, float *minX, float *maxX, float *minY, float *maxY,
                                   float *minZ, float *maxZ, int n, int m, SubDomainKeyTree *s, int *procCounter,
                                   int *procCounterTemp, bool timing) {

    float elapsedTime = 0.f;
    if (timing) {
        hipEvent_t start_t, stop_t; // used for timing
        hipEventCreate(&start_t);
        hipEventCreate(&stop_t);
        hipEventRecord(start_t, 0);

    particlesPerProcessKernel<<< gridSize, blockSize >>>(x, y, z, mass, count, start, child, index,
                                                   minX, maxX, minY, maxY, minZ, maxZ, n, m, s, procCounter,
                                                   procCounterTemp);

        hipEventRecord(stop_t, 0);
        hipEventSynchronize(stop_t);
        hipEventElapsedTime(&elapsedTime, start_t, stop_t);
        hipEventDestroy(start_t);
        hipEventDestroy(stop_t);
    }
    else {
        particlesPerProcessKernel<<< gridSize, blockSize >>>(x, y, z, mass, count, start, child, index,
                                                             minX, maxX, minY, maxY, minZ, maxZ, n, m, s, procCounter,
                                                             procCounterTemp);
    }
    return elapsedTime;

}

float KernelsWrapper::sortParticlesProc(float *x, float *y, float *z, float *mass, int *count, int *start,
                       int *child, int *index, float *minX, float *maxX, float *minY, float *maxY,
                       float *minZ, float *maxZ, int n, int m, SubDomainKeyTree *s, int *procCounter,
                       int *procCounterTemp, int *sortArray, bool timing) {

    float elapsedTime = 0.f;
    if (timing) {
        hipEvent_t start_t, stop_t; // used for timing
        hipEventCreate(&start_t);
        hipEventCreate(&stop_t);
        hipEventRecord(start_t, 0);

        sortParticlesProcKernel<<< gridSize, blockSize >>>(x, y, z, mass, count, start, child, index,
                                                         minX, maxX, minY, maxY, minZ, maxZ, n, m, s,
                                                         procCounter, procCounterTemp, sortArray);
        hipEventRecord(stop_t, 0);
        hipEventSynchronize(stop_t);
        hipEventElapsedTime(&elapsedTime, start_t, stop_t);
        hipEventDestroy(start_t);
        hipEventDestroy(stop_t);
    }
    else {
        sortParticlesProcKernel<<< gridSize, blockSize >>>(x, y, z, mass, count, start, child, index,
                                                           minX, maxX, minY, maxY, minZ, maxZ, n, m, s,
                                                           procCounter, procCounterTemp, sortArray);
    }
    return elapsedTime;
}

float KernelsWrapper::copyArray(float *targetArray, float *sourceArray, int n, bool timing) {

    float elapsedTime = 0.f;
    if (timing) {
        hipEvent_t start_t, stop_t; // used for timing
        hipEventCreate(&start_t);
        hipEventCreate(&stop_t);
        hipEventRecord(start_t, 0);

        copyArrayKernel<<<gridSize, blockSize>>>(targetArray, sourceArray, n);

        hipEventRecord(stop_t, 0);
        hipEventSynchronize(stop_t);
        hipEventElapsedTime(&elapsedTime, start_t, stop_t);
        hipEventDestroy(start_t);
        hipEventDestroy(stop_t);
    }
    else {
        copyArrayKernel<<<gridSize, blockSize>>>(targetArray, sourceArray, n);
    }
    return elapsedTime;
}

float KernelsWrapper::resetFloatArray(float *array, float value, int n, bool timing) {

    float elapsedTime = 0.f;
    if (timing) {
        hipEvent_t start_t, stop_t; // used for timing
        hipEventCreate(&start_t);
        hipEventCreate(&stop_t);
        hipEventRecord(start_t, 0);

        resetFloatArrayKernel<<<gridSize, blockSize>>>(array, value, n);

        hipEventRecord(stop_t, 0);
        hipEventSynchronize(stop_t);
        hipEventElapsedTime(&elapsedTime, start_t, stop_t);
        hipEventDestroy(start_t);
        hipEventDestroy(stop_t);
    }
    else {
        resetFloatArrayKernel<<<gridSize, blockSize>>>(array, value, n);
    }
    return elapsedTime;

}

float KernelsWrapper::reorderArray(float *array, float *tempArray, SubDomainKeyTree *s,
                  int *procCounter, int *receiveLengths, bool timing) {

    float elapsedTime = 0.f;
    if (timing) {
        hipEvent_t start_t, stop_t; // used for timing
        hipEventCreate(&start_t);
        hipEventCreate(&stop_t);
        hipEventRecord(start_t, 0);

        //reorderArrayKernel<<<gridSize, blockSize>>>(array, tempArray, s, procCounter, receiveLengths);

        hipEventRecord(stop_t, 0);
        hipEventSynchronize(stop_t);
        hipEventElapsedTime(&elapsedTime, start_t, stop_t);
        hipEventDestroy(start_t);
        hipEventDestroy(stop_t);
    }
    else {
        //reorderArrayKernel<<<gridSize, blockSize>>>(array, tempArray, s, procCounter, receiveLengths);
    }
    return elapsedTime;
}

float KernelsWrapper::sendParticles(float *x, float *y, float *z, float *mass, int *count, int *start,
                   int *child, int *index, float *minX, float *maxX, float *minY, float *maxY,
                   float *minZ, float *maxZ, int n, int m, SubDomainKeyTree *s, int *procCounter,
                   float *tempArray, int *sortArray, int *sortArrayOut, bool timing) {

    float elapsedTime = 0.f;
    if (timing) {
        hipEvent_t start_t, stop_t; // used for timing
        hipEventCreate(&start_t);
        hipEventCreate(&stop_t);
        hipEventRecord(start_t, 0);

        sendParticlesKernel<<< 1, 1/*gridSize, blockSize*/ >>>(x, y, z, mass, count, start, child, index,
                                                   minX, maxX, minY, maxY, minZ, maxZ, n, m, s, procCounter,
                                                   tempArray, sortArray, sortArrayOut);


        hipEventRecord(stop_t, 0);
        hipEventSynchronize(stop_t);
        hipEventElapsedTime(&elapsedTime, start_t, stop_t);
        hipEventDestroy(start_t);
        hipEventDestroy(stop_t);
    }
    else {
        sendParticlesKernel<<< 1, 1/*gridSize, blockSize*/ >>>(x, y, z, mass, count, start, child, index,
                                                               minX, maxX, minY, maxY, minZ, maxZ, n, m, s, procCounter,
                                                               tempArray, sortArray, sortArrayOut);
    }

    //printf("Elapsed time for sorting: %f\n", elapsedTime);
    return elapsedTime;

}

float KernelsWrapper::buildTree(float *x, float *y, float *z, float *mass, int *count, int *start,
                        int *child, int *index, float *minX, float *maxX, float *minY, float *maxY,
                        float *minZ, float *maxZ, int n, int m, bool timing) {

    float elapsedTime = 0.f;
    if (timing) {
        hipEvent_t start_t, stop_t; // used for timing
        hipEventCreate(&start_t);
        hipEventCreate(&stop_t);
        hipEventRecord(start_t, 0);

        buildTreeKernel<<< gridSize, blockSize >>>(x, y, z, mass, count, start, child, index,
                minX, maxX, minY, maxY, minZ, maxZ, n, m);

        hipEventRecord(stop_t, 0);
        hipEventSynchronize(stop_t);
        hipEventElapsedTime(&elapsedTime, start_t, stop_t);
        hipEventDestroy(start_t);
        hipEventDestroy(stop_t);
    }
    else {
        buildTreeKernel<<< gridSize, blockSize >>>(x, y, z, mass, count, start, child, index,
                                                   minX, maxX, minY, maxY, minZ, maxZ, n, m);
    }
    return elapsedTime;

}

float KernelsWrapper::getParticleKey(float *x, float *y, float *z, float *minX, float *maxX, float *minY, float *maxY,
                    float *minZ, float *maxZ, unsigned long *key, int maxLevel, int n, SubDomainKeyTree *s, bool timing) {

    float elapsedTime = 0.f;
    if (timing) {
        hipEvent_t start_t, stop_t; // used for timing
        hipEventCreate(&start_t);
        hipEventCreate(&stop_t);
        hipEventRecord(start_t, 0);

        getParticleKeyKernel<<< gridSize, blockSize >>>(x, y, z, minX, maxX, minY, maxY, minZ, maxZ, 0UL, 21, n, s);

        hipEventRecord(stop_t, 0);
        hipEventSynchronize(stop_t);
        hipEventElapsedTime(&elapsedTime, start_t, stop_t);
        hipEventDestroy(start_t);
        hipEventDestroy(stop_t);
    }
    else {
        getParticleKeyKernel<<< gridSize, blockSize >>>(x, y, z, minX, maxX, minY, maxY, minZ, maxZ, 0UL, 21, n, s);
    }
    return elapsedTime;

}

float KernelsWrapper::traverseIterative(float *x, float *y, float *z, float *mass, int *child, int n, int m,
                       SubDomainKeyTree *s, int maxLevel, bool timing) {

    float elapsedTime = 0.f;
    if (timing) {
        hipEvent_t start_t, stop_t; // used for timing
        hipEventCreate(&start_t);
        hipEventCreate(&stop_t);
        hipEventRecord(start_t, 0);

        traverseIterativeKernel<<< 1, 1 >>>(x, y, z, mass, child, n, m, s, maxLevel);

        hipEventRecord(stop_t, 0);
        hipEventSynchronize(stop_t);
        hipEventElapsedTime(&elapsedTime, start_t, stop_t);
        hipEventDestroy(start_t);
        hipEventDestroy(stop_t);
    }
    else {
        traverseIterativeKernel<<< 1, 1 >>>(x, y, z, mass, child, n, m, s, maxLevel);
    }
    return elapsedTime;

}

void KernelsWrapper::createDomainList(float *x, float *y, float *z, float *mass, int *child, int n,
                                      SubDomainKeyTree *s, int maxLevel) {
    //createDomainListKernel<<< gridSize, blockSize >>>(x, y, z, mass, child, n, s, maxLevel);
};

/*void KernelsWrapper::createDomainList(float *x, float *y, float *z, float *mass, float *minX, float *maxX,
                                            float *minY, float *maxY, float *minZ, float *maxZ, int *child, int n,
                                            SubDomainKeyTree *s, int maxLevel) {
    createDomainListKernel<<< 1, 1 >>>(x, y, z, mass, minX, maxX, minY, maxY, minZ, maxZ,
                                                      child, n, s, maxLevel);
};*/

float KernelsWrapper::createDomainList(SubDomainKeyTree *s, int maxLevel, unsigned long *domainListKeys, int *levels,
                                      int *index, bool timing) {

    float elapsedTime = 0.f;
    if (timing) {
        hipEvent_t start_t, stop_t; // used for timing
        hipEventCreate(&start_t);
        hipEventCreate(&stop_t);
        hipEventRecord(start_t, 0);

        createDomainListKernel<<<1, 1>>>(s, maxLevel, domainListKeys, levels, index);

        hipEventRecord(stop_t, 0);
        hipEventSynchronize(stop_t);
        hipEventElapsedTime(&elapsedTime, start_t, stop_t);
        hipEventDestroy(start_t);
        hipEventDestroy(stop_t);
    }
    else {
        createDomainListKernel<<<1, 1>>>(s, maxLevel, domainListKeys, levels, index);
    }
    return elapsedTime;
}

float KernelsWrapper::centreOfMass(float *x, float *y, float *z, float *mass, int *index, int n, bool timing) {

    float elapsedTime = 0.f;
    if (timing) {
        hipEvent_t start_t, stop_t; // used for timing
        hipEventCreate(&start_t);
        hipEventCreate(&stop_t);
        hipEventRecord(start_t, 0);

        centreOfMassKernel<<< gridSize, blockSize >>>(x, y, z, mass, index, n);

        hipEventRecord(stop_t, 0);
        hipEventSynchronize(stop_t);
        hipEventElapsedTime(&elapsedTime, start_t, stop_t);
        hipEventDestroy(start_t);
        hipEventDestroy(stop_t);
    }
    else {
        centreOfMassKernel<<< gridSize, blockSize >>>(x, y, z, mass, index, n);
    }
    return elapsedTime;

}

float KernelsWrapper::sort(int *count, int *start, int *sorted, int *child, int *index, int n, bool timing) {

    float elapsedTime = 0.f;
    if (timing) {
        hipEvent_t start_t, stop_t; // used for timing
        hipEventCreate(&start_t);
        hipEventCreate(&stop_t);
        hipEventRecord(start_t, 0);

        sortKernel<<< gridSize, blockSize>>>(count, start, sorted, child, index, n);

        hipEventRecord(stop_t, 0);
        hipEventSynchronize(stop_t);
        hipEventElapsedTime(&elapsedTime, start_t, stop_t);
        hipEventDestroy(start_t);
        hipEventDestroy(stop_t);
    }
    else {
        sortKernel<<< gridSize, blockSize>>>(count, start, sorted, child, index, n);
    }
    return elapsedTime;

}

float KernelsWrapper::computeForces(float *x, float *y, float *z, float *vx, float *vy, float *vz,
                            float *ax, float *ay, float *az, float *mass, int *sorted, int *child,
                            float *minX, float *maxX, int n, float g, bool timing) {

    float elapsedTime = 0.f;
    if (timing) {
        hipEvent_t start_t, stop_t; // used for timing
        hipEventCreate(&start_t);
        hipEventCreate(&stop_t);
        hipEventRecord(start_t, 0);

        computeForcesKernel<<<gridSize, blockSize, (sizeof(float)+sizeof(int))*stackSize*blockSizeInt/warp>>>(x, y, z, vx, vy, vz, ax, ay, az,
                mass, sorted, child, minX, maxX, n, g, blockSizeInt, warp, stackSize);

        hipEventRecord(stop_t, 0);
        hipEventSynchronize(stop_t);
        hipEventElapsedTime(&elapsedTime, start_t, stop_t);
        hipEventDestroy(start_t);
        hipEventDestroy(stop_t);
    }
    else {
        computeForcesKernel<<<gridSize, blockSize, (sizeof(float)+sizeof(int))*stackSize*blockSizeInt/warp>>>(x, y, z, vx, vy, vz, ax, ay, az,
                                                     mass, sorted, child, minX, maxX, n, g, blockSizeInt, warp, stackSize);
    }
    return elapsedTime;

}

float KernelsWrapper::update(float *x, float *y, float *z, float *vx, float *vy, float *vz,
                    float *ax, float *ay, float *az, int n, float dt, float d, bool timing) {

    float elapsedTime = 0.f;
    if (timing) {
        hipEvent_t start_t, stop_t; // used for timing
        hipEventCreate(&start_t);
        hipEventCreate(&stop_t);
        hipEventRecord(start_t, 0);

        updateKernel<<< gridSize, blockSize >>>(x, y, z, vx, vy, vz, ax, ay, az, n, dt, d);

        hipEventRecord(stop_t, 0);
        hipEventSynchronize(stop_t);
        hipEventElapsedTime(&elapsedTime, start_t, stop_t);
        hipEventDestroy(start_t);
        hipEventDestroy(stop_t);
    }
    else {
        updateKernel<<< gridSize, blockSize >>>(x, y, z, vx, vy, vz, ax, ay, az, n, dt, d);
    }
    return elapsedTime;

}

void KernelsWrapper::collectSendIndices(int *sendIndices, float *entry, float *tempArray, int *domainListCounter,
                                        int sendCount) {

    float elapsedTime = 0.f;
    if (timing) {
        hipEvent_t start_t, stop_t; // used for timing
        hipEventCreate(&start_t);
        hipEventCreate(&stop_t);
        hipEventRecord(start_t, 0);

        collectSendIndicesKernel<<< gridSize, blockSize >>>(sendIndices, entry, tempArray, domainListCounter,
                                                            sendCount);

        hipEventRecord(stop_t, 0);
        hipEventSynchronize(stop_t);
        hipEventElapsedTime(&elapsedTime, start_t, stop_t);
        hipEventDestroy(start_t);
        hipEventDestroy(stop_t);
    }
    else {
        collectSendIndicesKernel<<< gridSize, blockSize >>>(sendIndices, entry, tempArray, domainListCounter,
                                                            sendCount);
    }
    return elapsedTime;

}

