/**
 * Wrapping CUDA Kernel functions.
 */

#include "../include/KernelsWrapper.cuh"

/*
dim3 gridSize  = 1024; //2048; //1024; //512;
dim3 blockSize = 256; //256;
 */

KernelsWrapper::KernelsWrapper() {
    gridSize = 0;
    blockSize = 0;
    blockSizeInt = 0;
    warp = 0;
    stackSize = 0;
}

KernelsWrapper::KernelsWrapper(SimulationParameters p) {
    gridSize = p.gridSize;
    blockSize = p.blockSize;
    blockSizeInt = p.blockSize;
    warp = p.warp;
    stackSize = p.stackSize;
}

float KernelsWrapper::resetArrays(int *mutex, float *x, float *y, float *z, float *mass, int *count,
                          int *start, int *sorted, int *child, int *index, float *minX, float *maxX, float *minY, float *maxY,
                          float *minZ, float *maxZ, int n, int m, bool timing) {

    float elapsedTime = 0.f;
    if (timing) {
        hipEvent_t start_t, stop_t; // used for timing
        hipEventCreate(&start_t);
        hipEventCreate(&stop_t);
        hipEventRecord(start_t, 0);

        resetArraysKernel<<< gridSize, blockSize >>>(mutex, x, y, z, mass, count, start, sorted, child, index,
                minX, maxX, minY, maxY, minZ, maxZ, n, m);

        hipEventRecord(stop_t, 0);
        hipEventSynchronize(stop_t);
        hipEventElapsedTime(&elapsedTime, start_t, stop_t);
        hipEventDestroy(start_t);
        hipEventDestroy(stop_t);
    }
    else {
        resetArraysKernel<<< gridSize, blockSize >>>(mutex, x, y, z, mass, count, start, sorted, child, index,
                                                     minX, maxX, minY, maxY, minZ, maxZ, n, m);
    }
    return elapsedTime;

}

float KernelsWrapper::computeBoundingBox(int *mutex, float *x, float *y, float *z, float *minX,
                                 float *maxX, float *minY, float *maxY, float *minZ, float *maxZ, int n, bool timing) {

    float elapsedTime = 0.f;
    if (timing) {
        hipEvent_t start_t, stop_t; // used for timing
        hipEventCreate(&start_t);
        hipEventCreate(&stop_t);
        hipEventRecord(start_t, 0);

        computeBoundingBoxKernel<<< gridSize, blockSize, 6*sizeof(float)*blockSizeInt >>>(mutex, x, y, z, minX, maxX, minY, maxY, minZ, maxZ, n, blockSizeInt);

        hipEventRecord(stop_t, 0);
        hipEventSynchronize(stop_t);
        hipEventElapsedTime(&elapsedTime, start_t, stop_t);
        hipEventDestroy(start_t);
        hipEventDestroy(stop_t);
    }
    else {
        computeBoundingBoxKernel<<< gridSize, blockSize, 6*sizeof(float)*blockSizeInt >>>(mutex, x, y, z, minX, maxX, minY, maxY, minZ, maxZ, n, blockSizeInt);
    }
    return elapsedTime;

}

float KernelsWrapper::buildTree(float *x, float *y, float *z, float *mass, int *count, int *start,
                        int *child, int *index, float *minX, float *maxX, float *minY, float *maxY,
                        float *minZ, float *maxZ, int n, int m, bool timing) {

    float elapsedTime = 0.f;
    if (timing) {
        hipEvent_t start_t, stop_t; // used for timing
        hipEventCreate(&start_t);
        hipEventCreate(&stop_t);
        hipEventRecord(start_t, 0);

        buildTreeKernel<<< gridSize, blockSize >>>(x, y, z, mass, count, start, child, index,
                minX, maxX, minY, maxY, minZ, maxZ, n, m);

        hipEventRecord(stop_t, 0);
        hipEventSynchronize(stop_t);
        hipEventElapsedTime(&elapsedTime, start_t, stop_t);
        hipEventDestroy(start_t);
        hipEventDestroy(stop_t);
    }
    else {
        buildTreeKernel<<< gridSize, blockSize >>>(x, y, z, mass, count, start, child, index,
                                                   minX, maxX, minY, maxY, minZ, maxZ, n, m);
    }
    return elapsedTime;

}

void KernelsWrapper::getParticleKey(float *x, float *y, float *z, float *minX, float *maxX, float *minY, float *maxY,
                    float *minZ, float *maxZ, unsigned long *key, int maxLevel, int n) {

    getParticleKeyKernel<<< gridSize, blockSize >>>(x, y, z, minX, maxX, minY, maxY, minZ, maxZ, 0UL, 21, n);

}

float KernelsWrapper::centreOfMass(float *x, float *y, float *z, float *mass, int *index, int n, bool timing) {

    float elapsedTime = 0.f;
    if (timing) {
        hipEvent_t start_t, stop_t; // used for timing
        hipEventCreate(&start_t);
        hipEventCreate(&stop_t);
        hipEventRecord(start_t, 0);

        centreOfMassKernel<<< gridSize, blockSize >>>(x, y, z, mass, index, n);

        hipEventRecord(stop_t, 0);
        hipEventSynchronize(stop_t);
        hipEventElapsedTime(&elapsedTime, start_t, stop_t);
        hipEventDestroy(start_t);
        hipEventDestroy(stop_t);
    }
    else {
        centreOfMassKernel<<< gridSize, blockSize >>>(x, y, z, mass, index, n);
    }
    return elapsedTime;

}

float KernelsWrapper::sort(int *count, int *start, int *sorted, int *child, int *index, int n, bool timing) {

    float elapsedTime = 0.f;
    if (timing) {
        hipEvent_t start_t, stop_t; // used for timing
        hipEventCreate(&start_t);
        hipEventCreate(&stop_t);
        hipEventRecord(start_t, 0);

        sortKernel<<< gridSize, blockSize>>>(count, start, sorted, child, index, n);

        hipEventRecord(stop_t, 0);
        hipEventSynchronize(stop_t);
        hipEventElapsedTime(&elapsedTime, start_t, stop_t);
        hipEventDestroy(start_t);
        hipEventDestroy(stop_t);
    }
    else {
        sortKernel<<< gridSize, blockSize>>>(count, start, sorted, child, index, n);
    }
    return elapsedTime;

}

float KernelsWrapper::computeForces(float *x, float *y, float *z, float *vx, float *vy, float *vz,
                            float *ax, float *ay, float *az, float *mass, int *sorted, int *child,
                            float *minX, float *maxX, int n, float g, bool timing) {

    float elapsedTime = 0.f;
    if (timing) {
        hipEvent_t start_t, stop_t; // used for timing
        hipEventCreate(&start_t);
        hipEventCreate(&stop_t);
        hipEventRecord(start_t, 0);

        computeForcesKernel<<<gridSize, blockSize, (sizeof(float)+sizeof(int))*stackSize*blockSizeInt/warp>>>(x, y, z, vx, vy, vz, ax, ay, az,
                mass, sorted, child, minX, maxX, n, g, blockSizeInt, warp, stackSize);

        hipEventRecord(stop_t, 0);
        hipEventSynchronize(stop_t);
        hipEventElapsedTime(&elapsedTime, start_t, stop_t);
        hipEventDestroy(start_t);
        hipEventDestroy(stop_t);
    }
    else {
        computeForcesKernel<<<gridSize, blockSize, (sizeof(float)+sizeof(int))*stackSize*blockSizeInt/warp>>>(x, y, z, vx, vy, vz, ax, ay, az,
                                                     mass, sorted, child, minX, maxX, n, g, blockSizeInt, warp, stackSize);
    }
    return elapsedTime;

}

float KernelsWrapper::update(float *x, float *y, float *z, float *vx, float *vy, float *vz,
                    float *ax, float *ay, float *az, int n, float dt, float d, bool timing) {

    float elapsedTime = 0.f;
    if (timing) {
        hipEvent_t start_t, stop_t; // used for timing
        hipEventCreate(&start_t);
        hipEventCreate(&stop_t);
        hipEventRecord(start_t, 0);

        updateKernel<<< gridSize, blockSize >>>(x, y, z, vx, vy, vz, ax, ay, az, n, dt, d);

        hipEventRecord(stop_t, 0);
        hipEventSynchronize(stop_t);
        hipEventElapsedTime(&elapsedTime, start_t, stop_t);
        hipEventDestroy(start_t);
        hipEventDestroy(stop_t);
    }
    else {
        updateKernel<<< gridSize, blockSize >>>(x, y, z, vx, vy, vz, ax, ay, az, n, dt, d);
    }
    return elapsedTime;

}

